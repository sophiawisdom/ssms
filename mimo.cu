#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <builtin_types.h>

#include <vector>

hipModule_t ptx_module;
hipFunction_t kernel_function;
hipDevice_t   device;
hipCtx_t  context;
int major = 0, minor = 0;


__attribute__((constructor))
static void initialize_cuda() {
    hipError_t err = hipInit(0);

    err = hipCtxCreate(&context, 0, device);

    hipDeviceGet(&device, 0);
    hipDeviceComputeCapability(&major, &minor, device);
    err = hipModuleLoad(&ptx_module, "mimo_done.o");
    if (err != hipSuccess) {
        hipError_t error = hipGetLastError();
        const char *name = hipGetErrorName(error);
        const char *string = hipGetErrorString(error);
        fprintf(stderr, "* Error loading PTX module. Error name \"%s\" string \"%s\" err is %d\n", name, string, err);
        exit(-1);
    }

    err = hipModuleGetFunction(&kernel_function, ptx_module, "mimo_kernel");
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get function");
    }
}

template <typename scalar_t>
__global__ void mimo_cuda_forward_kernel(
    scalar_t * __restrict__ u,
    scalar_t * __restrict__ a,
    scalar_t * __restrict__ b,
    scalar_t * __restrict__ c,
    scalar_t * __restrict__ d,
    scalar_t * __restrict__ out
) {
    out[threadIdx.x + blockIdx.x * blockDim.x] = 6;
}

torch::Tensor mimo_cuda_forward(
    torch::Tensor u,
    torch::Tensor a,
    torch::Tensor b,
    torch::Tensor c,
    torch::Tensor d
) {

  const auto num_heads = a.size(0); // {N_HEADS, STATE_SIZE}
  const auto batch_size = u.size(1);

  auto output = torch::zeros_like(u);

  void * argBuffer[6];
  int argBufferSize = 6*8; // 6 pointers
  argBuffer[0] = u.data_ptr();
  argBuffer[1] = a.data_ptr();
  argBuffer[2] = b.data_ptr();
  argBuffer[3] = c.data_ptr();
  argBuffer[4] = d.data_ptr();
  argBuffer[5] = output.data_ptr();
  for (int i = 0; i < 6; i++) {
    printf("argBuffer for #%d is %p\n", i, argBuffer[i]);
  }

  void *config[] = {
    HIP_LAUNCH_PARAM_BUFFER_POINTER, argBuffer,
    HIP_LAUNCH_PARAM_BUFFER_SIZE,    &argBufferSize,
    HIP_LAUNCH_PARAM_END,
  };
  printf("about to hipModuleLaunchKernel\n");
  int error = hipModuleLaunchKernel(kernel_function,
  1, 1, 1, // grid x, y, z
  32, 1, 1, // block x, y, z
  0, 0, NULL, config);
  if (error != hipSuccess) {
    hipError_t lastErr = hipGetLastError();
    const char *name = hipGetErrorName(lastErr);
    const char *string = hipGetErrorString(lastErr);
    fprintf(stderr, "* Error with hipModuleLaunchKernel. Error name \"%s\" string \"%s\" err is %d\n", name, string, error);
  }
  printf("hipModuleLaunchKernel result is %d\n", error);
  // mimo_cuda_forward_kernel<<<num_heads, 32>>>(u.data<float>(), a.data<float>(), b.data<float>(), c.data<float>(), d.data<float>(), output.data<float>());

  return output;
}