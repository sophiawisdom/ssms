#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <builtin_types.h>

#include <vector>

hipModule_t ptx_module;
hipFunction_t kernel_function;
hipDevice_t   device;
hipCtx_t  context;
int major = 0, minor = 0;

// #define DEBUG

__attribute__((constructor))
static void initialize_cuda() {
    hipError_t err = hipInit(0);

    err = hipCtxCreate(&context, 0, device);

    hipDeviceGet(&device, 0);
    hipDeviceComputeCapability(&major, &minor, device);
    err = hipModuleLoad(&ptx_module, "siso.o");
    if (err != hipSuccess) {
        hipError_t error = hipGetLastError();
        const char *name = hipGetErrorName(error);
        const char *string = hipGetErrorString(error);
        fprintf(stderr, "* Error loading PTX module. Error name \"%s\" string \"%s\" err is %d\n", name, string, err);
        exit(-1);
    }

    err = hipModuleGetFunction(&kernel_function, ptx_module, "siso_kernel");
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get function");
    }

    printf("SISO initialized!");
}

unsigned int old_sequence_length = 0;

torch::Tensor siso_cuda_forward(
    torch::Tensor u,
    torch::Tensor a,
    torch::Tensor b,
    torch::Tensor c,
    unsigned int sequence_length
) {

  const auto num_heads = a.size(0); // {N_HEADS, STATE_SIZE}

  auto output = torch::zeros_like(u);

  assert(num_heads % 4 == 0);

  // printf("a sizes 0 is %d\n", a.sizes()[0]);
  unsigned int n_heads = a.sizes()[0];
  
  void * argBuffer[6];
  int argBufferSize = 5*8 + 4; // 5 pointers and an int
  argBuffer[0] = u.data_ptr();
  argBuffer[1] = a.data_ptr();
  argBuffer[2] = b.data_ptr();
  argBuffer[3] = c.data_ptr();
  argBuffer[4] = output.data_ptr();
  int *argBufferView = (int *)&argBuffer;
  argBufferView[10] = sequence_length;

  if (old_sequence_length != sequence_length) {
#ifdef DEBUG
    printf("sequence_length %u\n", sequence_length);
#endif
    old_sequence_length = sequence_length;
  }

  for (int i = 0; i < sizeof(argBuffer)/sizeof(void *) + 1; i++) {
#ifdef DEBUG
    printf("argBuffer for #%d is %p\n", i, argBuffer[i]);
#endif
  }

  void *config[] = {
    HIP_LAUNCH_PARAM_BUFFER_POINTER, argBufferView,
    HIP_LAUNCH_PARAM_BUFFER_SIZE,    &argBufferSize,
    HIP_LAUNCH_PARAM_END,
  };
#ifdef DEBUG
  printf("about to hipModuleLaunchKernel, sequence_length is %d\n", sequence_length);
#endif
  int error = hipModuleLaunchKernel(kernel_function,
  num_heads/4, 1, 1, // grid x, y, z
  8, 4, 1, // block x, y, z
  0, 0, NULL, config);
  if (error != hipSuccess) {
    hipError_t lastErr = hipGetLastError();
    const char *name = hipGetErrorName(lastErr);
    const char *string = hipGetErrorString(lastErr);
    fprintf(stderr, "* Error with hipModuleLaunchKernel. Error name \"%s\" string \"%s\" err is %d\n", name, string, error);
  }
#ifdef DEBUG
  printf("hipModuleLaunchKernel result is %d\n", error);
#endif

  return output;
}