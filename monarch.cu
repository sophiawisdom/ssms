#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <builtin_types.h>
#include <mma.h>

#include <vector>

hipModule_t ptx_module;
hipFunction_t kernel_function;
hipDevice_t   device;
hipCtx_t  context;
int major = 0, minor = 0;

using namespace nvcuda;

__global__ void monarch_impl(__hip_bfloat16 *x_ptr, __hip_bfloat16 *w1_ptr, float *out_ptr) {
  // ROOT_N = 128 (N=16384), BATCH_SIZE = 16
  int warp = threadIdx.x / 32;

  wmma::fragment<wmma::matrix_a, 16, 16, 16, __hip_bfloat16, wmma::col_major> x_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, __hip_bfloat16, wmma::row_major> w1_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> out_frag;

  // xs are the same for every warp, weights are different
  __hip_bfloat16 *per_warp_w1 = w1_ptr + warp*32;

  wmma::fill_fragment(out_frag, 0.0f); // initialize out to 0

  for (int i = 0; i < 8; i++) {
      wmma::load_matrix_sync(x_frag, x_ptr + (i * 16 * sizeof(__hip_bfloat16)), 128); // load which 16x16 chunk
      wmma::load_matrix_sync(w1_frag, per_warp_w1 + (i * 2048 * sizeof(__hip_bfloat16)), 128);
      wmma::mma_sync(out_frag, x_frag, w1_frag, out_frag);
  }

  float *per_warp_out = out_ptr + warp*32;
  wmma::store_matrix_sync(per_warp_out, out_frag, 128, wmma::mem_row_major);
}

__attribute__((constructor))
static void initialize_cuda() {
    hipError_t err = hipInit(0);

    err = hipCtxCreate(&context, 0, device);

    hipDeviceGet(&device, 0);
    hipDeviceComputeCapability(&major, &minor, device);
    err = hipModuleLoad(&ptx_module, "monarch.o");
    if (err != hipSuccess) {
        hipError_t error = hipGetLastError();
        const char *name = hipGetErrorName(error);
        const char *string = hipGetErrorString(error);
        fprintf(stderr, "* Error loading PTX module. Error name \"%s\" string \"%s\" err is %d\n", name, string, err);
        exit(-1);
    }

    err = hipModuleGetFunction(&kernel_function, ptx_module, "monarch_kernel");
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get function\n");
    }

    printf("Monarch initialized!\n");
}

torch::Tensor monarch_cuda_forward(
    torch::Tensor x,
    torch::Tensor w1_bfly
) {
  auto output = torch::zeros_like(x);

  unsigned int root_n = w1_bfly.sizes()[0];

  // monarch_impl<<<256, root_n>>>((__hip_bfloat16 *)x.data_ptr(), (__hip_bfloat16 *) w1_bfly.data_ptr(), (float *) out.data_ptr());
  // return out;

  void * argBuffer[3];
  int argBufferSize = sizeof(argBuffer);
  argBuffer[0] = x.data_ptr();
  argBuffer[1] = w1_bfly.data_ptr();
  argBuffer[2] = output.data_ptr();

  // printf("argBufferSize %d\n", argBufferSize);

  for (int i = 0; i < sizeof(argBuffer)/sizeof(void *) + 1; i++) {
#ifdef DEBUG
    printf("argBuffer for #%d is %p\n", i, argBuffer[i]);
#endif
  }

  void *config[] = {
    HIP_LAUNCH_PARAM_BUFFER_POINTER, argBuffer,
    HIP_LAUNCH_PARAM_BUFFER_SIZE,    &argBufferSize,
    HIP_LAUNCH_PARAM_END,
  };
#ifdef DEBUG
  printf("about to hipModuleLaunchKernel, sequence_length is %d\n", sequence_length);
#endif
  // printf("launching with %d grid\n", root_n);
  int error = hipModuleLaunchKernel(kernel_function,
  // root_n, 1, 1, // grid x, y, z
  1, 1, 1,
  // 32, 8, 1, // block x, y, z
  32, 8, 1,
  0, 0, NULL, config);
  if (error != hipSuccess) {
    hipError_t lastErr = hipGetLastError();
    const char *name = hipGetErrorName(lastErr);
    const char *string = hipGetErrorString(lastErr);
    fprintf(stderr, "* Error with hipModuleLaunchKernel. Error name \"%s\" string \"%s\" err is %d\n", name, string, error);
  }
#ifdef DEBUG
  printf("hipModuleLaunchKernel result is %d\n", error);
#endif

  return output;
}
