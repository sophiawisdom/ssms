#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <builtin_types.h>

#include <vector>

hipModule_t ptx_module;
hipFunction_t kernel_function;
hipDevice_t   device;
hipCtx_t  context;
int major = 0, minor = 0;

// #define DEBUG

__attribute__((constructor))
static void initialize_cuda() {
    hipError_t err = hipInit(0);

    err = hipCtxCreate(&context, 0, device);

    hipDeviceGet(&device, 0);
    hipDeviceComputeCapability(&major, &minor, device);
    err = hipModuleLoad(&ptx_module, "monarch.o");
    if (err != hipSuccess) {
        hipError_t error = hipGetLastError();
        const char *name = hipGetErrorName(error);
        const char *string = hipGetErrorString(error);
        fprintf(stderr, "* Error loading PTX module. Error name \"%s\" string \"%s\" err is %d\n", name, string, err);
        exit(-1);
    }

    err = hipModuleGetFunction(&kernel_function, ptx_module, "monarch_kernel");
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get function\n");
    }

    printf("Monarch initialized!\n");
}

torch::Tensor monarch_cuda_forward(
    torch::Tensor x,
    torch::Tensor w1_bfly
) {
  auto output = torch::zeros_like(x);

  unsigned int root_n = w1_bfly.sizes()[0];
  
  void * argBuffer[3];
  int argBufferSize = sizeof(argBuffer);
  argBuffer[0] = x.data_ptr();
  argBuffer[1] = w1_bfly.data_ptr();
  argBuffer[2] = output.data_ptr();

  printf("argBufferSize %d\n", argBufferSize);

  for (int i = 0; i < sizeof(argBuffer)/sizeof(void *) + 1; i++) {
#ifdef DEBUG
    printf("argBuffer for #%d is %p\n", i, argBuffer[i]);
#endif
  }

  void *config[] = {
    HIP_LAUNCH_PARAM_BUFFER_POINTER, argBuffer,
    HIP_LAUNCH_PARAM_BUFFER_SIZE,    &argBufferSize,
    HIP_LAUNCH_PARAM_END,
  };
#ifdef DEBUG
  printf("about to hipModuleLaunchKernel, sequence_length is %d\n", sequence_length);
#endif
  printf("launching with %d grid\n", root_n);
  int error = hipModuleLaunchKernel(kernel_function,
  root_n, 1, 1, // grid x, y, z
  32, 8, 1, // block x, y, z
  0, 0, NULL, config);
  if (error != hipSuccess) {
    hipError_t lastErr = hipGetLastError();
    const char *name = hipGetErrorName(lastErr);
    const char *string = hipGetErrorString(lastErr);
    fprintf(stderr, "* Error with hipModuleLaunchKernel. Error name \"%s\" string \"%s\" err is %d\n", name, string, error);
  }
#ifdef DEBUG
  printf("hipModuleLaunchKernel result is %d\n", error);
#endif

  return output;
}
